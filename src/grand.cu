#include "grand.h"

<<<<<<< HEAD
// ===================================================================================================
// Main driver test function.
//
// TO RUN:
// nvcc math.cu tensor.cu -o math
// compute-sanitizer .\math.exe (For debugging)
// ===================================================================================================
using namespace Grand;
int main()
{
    vector<vector<float>> data{{1, 2}, {3, 4}, {5, 6}};
    Tensor::Array a(data);
    Tensor::Array b(data);
    Tensor::Zeros c(a.tensor);

    // Add vectors in parallel.
    hipError_t cudaStatus = add(c.tensor, a.tensor, b.tensor, 0);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "ERROR: Addition failed.\n");
        return 1;
    }

    // Output
    for (int i = 0; i < c.tensor.width*c.tensor.height; i++)
    {
        cout << "C: " << c.tensor.data[i];
        cout << endl;
    }

=======
using namespace Grand;
int main()
{
    vector<vector<float>> data{{1, 2, 3}, {3, 4, 5}, {5, 6, 7}};
    Tensor::Array a(data);
>>>>>>> 249d41548953bdbc930f676b0b14330260921878
    return 0;
}