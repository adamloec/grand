#include "hip/hip_runtime.h"
#ifndef MATH_INCL
#define MATH_INCL
    #include <math.h>
    #include "math.h"
#endif

// ===================================================================================================
// NVIDIA COMPUTE CAPABILITY 8.6 SUPPORTED
// https://en.wikipedia.org/wiki/CUDA
//
// MAXIMUMS
// Threads per block = 1024
// Grids = 128
// Grid dimensions = (x, y, z)
//
//
// EXAMPLE KERNEL CALL
// kernel<<<ceil(n/256), 256>>>(args); //// <<<BLOCKS, THREADS PER BLOCK>>> n = flattened size of tensor
//
// EXAMPLE ERROR CHECKING
// if (err != hipSuccess)
// { 
//    printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
// }
//
// ===================================================================================================

namespace Grand
{
    // Add 2 tensor's kernel function.
    //
    // Tensor::Matrix c = Output tensor
    // Tensor::Matrix a/b = Input tensor's
    __global__ void addKernel(Tensor::Tensor c, Tensor::Tensor a, Tensor::Tensor b)
    {
        int i = blockDim.x * blockIdx.x + threadIdx.x;

        if (i < a.width*a.height)
        {
            c.data[i] = a.data[i] + b.data[i];
        }
    }

    // Add 2 tensor's function.
    //
    // Tensor::Array c = Output tensor
    // Tensor::Array a/b = Input tensor's
    hipError_t add(Tensor::Tensor c, Tensor::Tensor a, Tensor::Tensor b, int device=0)
    {
        Tensor::Tensor dev_a;
        Tensor::Tensor dev_b;
        Tensor::Tensor dev_c;
        size_t size;
        hipError_t cudaStatus;

        // CUDA device check
        cudaStatus = hipSetDevice(device);
        if (cudaStatus != hipSuccess) 
        {
            fprintf(stderr, "ERROR: Cuda enabled device {Device: %d} not found.\n", device);
            goto Error;
        }

        // Tensor input dimensions equality check
        if (a.width != b.width && a.height != b.height && c.width != a.width && c.height != a.height)
        {
            fprintf(stderr, "ERROR: Tensor dimensions do not match. A: {%d, %d} B: {%d, %d} C: {%d, %d}\n", a.width, a.height, b.width, b.height, c.width, c.height);
            goto Error;
        }

        // Data size (bytes)
        size = a.width * a.height * sizeof(float);

        // Initialize device tensor's width and height
        dev_a.width = a.width;
        dev_a.height = a.height;
        dev_b.width = b.width;
        dev_b.height = b.height;
        dev_c.width = c.width;
        dev_c.height = c.height;
        
        // Device memory allocation for input tensors
        hipMalloc(&dev_a.data, size);
        hipMalloc(&dev_b.data, size);

        // Copy input tensor's from host to device memory
        hipMemcpy(dev_a.data, a.data, size, hipMemcpyHostToDevice);
        hipMemcpy(dev_b.data, b.data, size, hipMemcpyHostToDevice);

        // Device memory allocation for output tensor
        hipMalloc(&dev_c.data, size);

        // Invoke kernel with specified kernel dimensions
        addKernel<<<ceil((a.width*a.height)/256.0), 256>>>(dev_c, dev_a, dev_b);

        // Kernel synchronize, checks for kernel errors
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) 
        {
            fprintf(stderr, "ERROR: Kernel synchronize failed: %d\n", cudaStatus);
            goto Error;
        }

        // Copy output tensor from device to host memory
        cudaStatus = hipMemcpy(c.data, dev_c.data, size, hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) 
        {
            fprintf(stderr, "ERROR: CUDAMEMCPY: %d\n", cudaStatus);
            goto Error;
        }

    Error:
        hipFree(dev_c.data);
        hipFree(dev_a.data);
        hipFree(dev_b.data);

        return cudaStatus;
    }
}


// ===================================================================================================
// Main driver test function.
//
// TO RUN:
// nvcc math.cu tensor.cu -o math
// compute-sanitizer .\math.exe (For debugging)
// ===================================================================================================
using namespace Grand;
int main()
{
    vector<vector<float>> data{{1, 2}, {3, 4}, {5, 6}};
    Tensor::Array a(data);
    Tensor::Array b(data);
    Tensor::Zeros c(a.tensor);

    // Add vectors in parallel.
    hipError_t cudaStatus = add(c.tensor, a.tensor, b.tensor);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "ERROR: Addition failed.\n");
        return 1;
    }

    // Output
    for (int i = 0; i < c.tensor.width*c.tensor.height; i++)
    {
        cout << "C: " << c.tensor.data[i];
        cout << endl;
    }

    return 0;
}
