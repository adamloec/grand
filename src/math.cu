#include "hip/hip_runtime.h"
#ifndef MATH_INCL
#define MATH_INCL
    #include <math.h>
    #include "math.h"
#endif

// ===================================================================================================
// NVIDIA COMPUTE CAPABILITY 8.6 SUPPORTED
// https://en.wikipedia.org/wiki/CUDA
//
// MAXIMUMS
// Threads per block = 1024
// Grids = 128
// Grid dimensions = (x, y, z)
//
//
// EXAMPLE KERNEL CALL
// kernel<<<ceil(n/256), 256>>>(args); //// <<<BLOCKS, THREADS PER BLOCK>>> n = flattened size of tensor
//
// EXAMPLE ERROR CHECKING
// if (err != hipSuccess)
// { 
//    printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
// }
//
// ===================================================================================================

namespace Grand
{
    // Add 2 tensor's kernel function.
    //
    // Tensor::Matrix c = Output tensor
    // Tensor::Matrix a/b = Input tensor's
    __global__ void addKernel(Tensor::Tensor c, Tensor::Tensor a, Tensor::Tensor b)
    {
        int i = blockDim.x * blockIdx.x + threadIdx.x;

        if (i < a.width*a.height)
        {
            c.data[i] = a.data[i] + b.data[i];
        }
    }

    // Dot product 2 tensor's kernel function.
    //
    // m * n matrix
    // n * k matrix
    // Tensor::Matrix c = Output tensor
    // Tensor::Matrix a/b = Input tensor's
    __global__ void dotKernel(Tensor::Tensor c, Tensor::Tensor a, Tensor::Tensor b)
    {
        int i = blockDim.x * blockIdx.x + threadIdx.x;
        int j = blockDim.y * blockIdx.y + threadIdx.y;

        if (i < a.width*a.height)
        {
            c.data[i] = a.data[i] + b.data[i];
        }
    }

    // Add 2 tensor's function.
    //
    // Tensor::Array c = Output tensor
    // Tensor::Array a/b = Input tensor's (m * n)
    hipError_t add(Tensor::Tensor c, Tensor::Tensor a, Tensor::Tensor b, int device=0)
    {
        Tensor::Tensor dev_a;
        Tensor::Tensor dev_b;
        Tensor::Tensor dev_c;
        size_t size;
        hipError_t cudaStatus;

        // CUDA device check
        cudaStatus = hipSetDevice(device);
        if (cudaStatus != hipSuccess) 
        {
            fprintf(stderr, "ERROR: Cuda enabled device {Device: %d} not found.\n", device);
            goto Error;
        }

        // Tensor input dimensions equality check
        if (a.width != b.width && a.height != b.height && c.width != a.width && c.height != a.height)
        {
            fprintf(stderr, "ERROR: Tensor dimensions do not match. A: {%d, %d} B: {%d, %d} C: {%d, %d}\n", a.width, a.height, b.width, b.height, c.width, c.height);
            goto Error;
        }

        // Data size (bytes)
        size = a.width * a.height * sizeof(float);

        // Initialize device tensor's width and height
        dev_a.width = a.width;
        dev_a.height = a.height;
        dev_b.width = b.width;
        dev_b.height = b.height;
        dev_c.width = c.width;
        dev_c.height = c.height;
        
        // Device memory allocation for input tensors
        hipMalloc(&dev_a.data, size);
        hipMalloc(&dev_b.data, size);

        // Copy input tensor's from host to device memory
        hipMemcpy(dev_a.data, a.data, size, hipMemcpyHostToDevice);
        hipMemcpy(dev_b.data, b.data, size, hipMemcpyHostToDevice);

        // Device memory allocation for output tensor
        hipMalloc(&dev_c.data, size);

        // Invoke kernel with specified kernel dimensions
        addKernel<<<ceil((a.width*a.height)/256.0), 256>>>(dev_c, dev_a, dev_b);

        // Kernel synchronize, checks for kernel errors
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) 
        {
            fprintf(stderr, "ERROR: Kernel synchronize failed: %d\n", cudaStatus);
            goto Error;
        }

        // Copy output tensor from device to host memory
        cudaStatus = hipMemcpy(c.data, dev_c.data, size, hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) 
        {
            fprintf(stderr, "ERROR: CUDAMEMCPY: %d\n", cudaStatus);
            goto Error;
        }

    Error:
        hipFree(dev_c.data);
        hipFree(dev_a.data);
        hipFree(dev_b.data);

        return cudaStatus;
    }

    // Multiply 2 tensor's function.
    //
    // Tensor::Array c = m * k output tensor
    // Tensor::Array a = m * n input tensor
    // Tensor::Array b = n * k input tensor
    hipError_t dot(Tensor::Tensor c, Tensor::Tensor a, Tensor::Tensor b, int device=0)
    {
        Tensor::Tensor dev_a;
        Tensor::Tensor dev_b;
        Tensor::Tensor dev_c;
        size_t size;
        hipError_t cudaStatus;

        // CUDA device check
        cudaStatus = hipSetDevice(device);
        if (cudaStatus != hipSuccess) 
        {
            fprintf(stderr, "ERROR: Cuda enabled device {Device: %d} not found.\n", device);
            goto Error;
        }

        

    Error:
        hipFree(dev_c.data);
        hipFree(dev_a.data);
        hipFree(dev_b.data);

        return cudaStatus;
    }
}