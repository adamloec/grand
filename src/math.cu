#include "hip/hip_runtime.h"
// ===================================================================================================
// Author: Adam Loeckle
// Date: 10/10/2022
// Description: Math source file.
// ===================================================================================================

#ifndef MATH_INCL
#define MATH_INCL
    #include <math.h>
    #include "math.h"
#endif

namespace Grand
{
    // Add 2 tensor's kernel function.
    //
    // Tensor::Matrix c = Output tensor
    // Tensor::Matrix a/b = Input tensor's
    __global__ void addKernel(Tensor::Tensor c, Tensor::Tensor a, Tensor::Tensor b)
    {
        int i = blockDim.x * blockIdx.x + threadIdx.x;

        if (i < a.width*a.height)
        {
            c.data[i] = a.data[i] + b.data[i];
        }
    }

    hipError_t add(Tensor::Tensor c, Tensor::Tensor a, Tensor::Tensor b, int device=0)
    {
        // Create device tensors.
        Tensor::Tensor dev_a;
        Tensor::Tensor dev_b;
        Tensor::Tensor dev_c;
        size_t size;
        hipError_t cudaStatus;

        // CUDA device check.
        cudaStatus = hipSetDevice(device);
        if (cudaStatus != hipSuccess) 
        {
            fprintf(stderr, "ERROR: Cuda enabled device {Device: %d} not found.\n", device);
            goto Error;
        }

        // Tensor input dimensions equality check.
        if (a.width != b.width && a.height != b.height && c.width != a.width && c.height != a.height)
        {
            fprintf(stderr, "ERROR: Tensor dimensions do not match. A: {%d, %d} B: {%d, %d} C: {%d, %d}\n", a.width, a.height, b.width, b.height, c.width, c.height);
            goto Error;
        }

        // Data size (bytes).
        size = a.width * a.height * sizeof(float);

        // Initialize device tensor's width and height.
        dev_a.width = a.width;
        dev_a.height = a.height;
        dev_b.width = b.width;
        dev_b.height = b.height;
        dev_c.width = c.width;
        dev_c.height = c.height;
        
        // Device memory allocation for input tensors.
        hipMalloc(&dev_a.data, size);
        hipMalloc(&dev_b.data, size);

        // Copy input tensor's from host to device memory.
        hipMemcpy(dev_a.data, a.data, size, hipMemcpyHostToDevice);
        hipMemcpy(dev_b.data, b.data, size, hipMemcpyHostToDevice);

        // Device memory allocation for output tensor.
        hipMalloc(&dev_c.data, size);

        // Invoke kernel with specified kernel dimensions.
        addKernel<<<ceil((a.width*a.height)/256.0), 256>>>(dev_c, dev_a, dev_b);

        // Kernel synchronize, checks for kernel errors.
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) 
        {
            fprintf(stderr, "ERROR: Kernel synchronize failed: %d\n", cudaStatus);
            goto Error;
        }

        // Copy output tensor from device to host memory.
        cudaStatus = hipMemcpy(c.data, dev_c.data, size, hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) 
        {
            fprintf(stderr, "ERROR: CUDAMEMCPY: %d\n", cudaStatus);
            goto Error;
        }

    // Error checking.
    Error:
        hipFree(dev_c.data);
        hipFree(dev_a.data);
        hipFree(dev_b.data);

        return cudaStatus;
    }
}


// ===================================================================================================
// Main driver test function.
//
// TO RUN:
// nvcc math.cu tensor.cu -o math
// compute-sanitizer .\math.exe (For debugging)
// ===================================================================================================
using namespace Grand;
int main()
{
    vector<vector<float>> data{{1, 2, 3}, {4, 5, 6}, {7, 8, 9}, {10, 11, 12}};
    Tensor::Array a(data);
    Tensor::Array b(data);
    Tensor::Zeros c(a.tensor);

    // Add vectors in parallel.
    hipError_t cudaStatus = add(c.tensor, a.tensor, b.tensor);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "ERROR: Addition failed.\n");
        return 1;
    }

    // Output
    for (int i = 0; i < c.tensor.width*c.tensor.height; i++)
    {
        cout << "C: " << c.tensor.data[i];
        cout << endl;
    }

    return 0;
}
