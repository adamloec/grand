#include "hip/hip_runtime.h"
#include "math.h"

__global__ void addKernel(Tensor c, Tensor a, Tensor b)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    
    c.data[i][j] = a.data[i][j] + b.data[i][j];
}

hipError_t add(Tensor c, Tensor a, Tensor b, int device=0)
{
    Tensor dev_a;
    Tensor dev_b;
    Tensor dev_c;
    size_t size;
    hipError_t cudaStatus;

    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(a.height / dimBlock.x, a.height / dimBlock.y);

    // CUDA device check
    cudaStatus = hipSetDevice(device);
    if (cudaStatus != hipSuccess) 
    {
        fprintf(stderr, "ERROR: Cuda enabled device {Device: %d} not found.\n", device);
        goto Error;
    }

    // Tensor input dimensions equality check
    if (a.width != b.width && a.height != b.height && c.width != a.width && c.height != a.height)
    {
        fprintf(stderr, "ERROR: Tensor dimensions do not match. A: {%d, %d} B: {%d, %d} C: {%d, %d}\n", a.width, a.height, b.width, b.height, c.width, c.height);
        goto Error;
    }

    // Constant width/height dimensions for Tensors
    size = a.width * a.height * sizeof(float);

    // Initialize input tensors and copy to memory
    dev_a.width = a.width;
    dev_a.height = a.height;
    hipMalloc(&dev_a.data, size);
    hipMemcpy(dev_a.data, a.data, size, hipMemcpyHostToDevice);

    dev_b.width = b.width;
    dev_b.height = b.height;
    hipMalloc(&dev_b.data, size);
    hipMemcpy(dev_b.data, b.data, size, hipMemcpyHostToDevice);

    // Initialize output tensor and copy to memory
    dev_c.width = c.width;
    dev_c.height = c.height;
    hipMalloc(&dev_c.data, size);

    // Generate kernel dimensions, invoke kernel
    addKernel<<<dimGrid, dimBlock>>>(dev_c, dev_a, dev_b);

    // Kernel synchronize, checks for kernel errors
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) 
    {
        fprintf(stderr, "ERROR: Kernel synchronize failed: %d\n", cudaStatus);
        goto Error;
    }

    // Read output tensor from memory
    cudaStatus = hipMemcpy(c.data, dev_c.data, size, hipMemcpyDeviceToHost);

Error:
    hipFree(dev_c.data);
    hipFree(dev_a.data);
    hipFree(dev_b.data);

    return cudaStatus;
}

int main()
{
    Tensor c;
    Tensor a;
    Tensor b;

    // Test data
    c.height = 2;
    c.width = 2;
    c.data = new float*[2];
    for (int i = 0; i < 2; i++)
    {
        c.data[i] = new float[2];
    }

    a.height = 2;
    a.width = 2;
    a.data = new float*[2];
    for (int i = 0; i < 2; i++)
    {
        a.data[i] = new float[2];
    }

    a.data[0][0] = 1.0;
    a.data[0][1] = 2.0;
    a.data[1][0] = 3.0;
    a.data[1][1] = 4.0;

    b.height = 2;
    b.width = 2;
    b.data = new float*[2];
    for (int i = 0; i < 2; i++)
    {
        b.data[i] = new float[2];
    }

    b.data[0][0] = 1.0;
    b.data[0][1] = 2.0;
    b.data[1][0] = 3.0;
    b.data[1][1] = 4.0;

    // Add vectors in parallel.
    hipError_t cudaStatus = add(c, a, b);
    if (cudaStatus != hipSuccess) 
    {
        fprintf(stderr, "ERROR: Addition failed.\n");
        return 1;
    }

    // Output
    for (int i = 0; i < 2; i++)
    {
        for (int j = 0; j < 2; j++)
        {
            cout << c.data[i][j];
        }
        cout << endl;
    }


    free(c.data);
    free(a.data);
    free(b.data);

    hipDeviceReset();

    return 0;
}