#include "hip/hip_runtime.h"


#include <stdio.h>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}

// Matrices are stored in row-major order:
// M(row, col) = *(M.elements + row * M.width + col)
typedef struct {
    int width;
    int height;
    float* elements;
} Matrix;

// Thread block size
#define BLOCK_SIZE 16

// Forward declaration of the matrix multiplication kernel
__global__ void MatMulKernel(const Matrix, const Matrix, Matrix);

// Matrix multiplication - Host code
// Matrix dimensions are assumed to be multiples of BLOCK_SIZE
void MatMul(const Matrix A, const Matrix B, Matrix C)
{
    // Load A and B to device memory
    Matrix d_A;
    d_A.width = A.width; d_A.height = A.height;
    size_t size = A.width * A.height * sizeof(float);
    hipMalloc(&d_A.elements, size);
    hipMemcpy(d_A.elements, A.elements, size,
               hipMemcpyHostToDevice);
    Matrix d_B;
    d_B.width = B.width; d_B.height = B.height;
    size = B.width * B.height * sizeof(float);
    hipMalloc(&d_B.elements, size);
    hipMemcpy(d_B.elements, B.elements, size,
               hipMemcpyHostToDevice);

    // Allocate C in device memory
    Matrix d_C;
    d_C.width = C.width; d_C.height = C.height;
    size = C.width * C.height * sizeof(float);
    hipMalloc(&d_C.elements, size);

    // Invoke kernel
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(B.width / dimBlock.x, A.height / dimBlock.y);
    MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);

    // Read C from device memory
    hipMemcpy(C.elements, d_C.elements, size,
               hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_A.elements);
    hipFree(d_B.elements);
    hipFree(d_C.elements);
}

// Matrix multiplication kernel called by MatMul()
__global__ void MatMulKernel(Matrix A, Matrix B, Matrix C)
{
    // Each thread computes one element of C
    // by accumulating results into Cvalue
    float Cvalue = 0;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    for (int e = 0; e < A.width; ++e)
        Cvalue += A.elements[row * A.width + e]
                * B.elements[e * B.width + col];
    C.elements[row * C.width + col] = Cvalue;
}

int main()
{
    Matrix a;
    Matrix b;
    Matrix c;

    a.width = 2;
    a.height = 2;
    b.width = 2;
    b.height = 2;
    c.width = 2;
    c.height = 2;

    

    hipDeviceReset();
    return 0;
}