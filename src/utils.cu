#include "hip/hip_runtime.h"
#include "utils.h"

__global__ void addKernal(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

hipError_t add(int *c, const int *a, const int *b, unsigned int size, int device=0)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    cudaStatus = hipSetDevice(device);
    if (cudaStatus != hipSuccess) 
    {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    Error:
        hipFree(dev_c);
        hipFree(dev_a);
        hipFree(dev_b);
        
        return cudaStatus;
}

int main()
{
    const int arraySize = 5;
    const int a[arraySize] = {1, 2, 3, 4, 5};
    const int b[arraySize] = {10, 20, 30, 40, 50};
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = add(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) 
    {
        fprintf(stderr, "ERROR: Addition failed.");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}