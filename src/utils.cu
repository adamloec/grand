#include "hip/hip_runtime.h"
#include "utils.h"

__global__ void addKernal(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

hipError_t add(int *c, const int *a, const int *b, unsigned int size, int device=0)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    cudaStatus = hipSetDevice(device);
    if (cudaStatus != hipSuccess) 
    {
        fprintf(stderr, "ERROR: Cuda enabled device {Device: %d} not found.\n", device);
        goto Error;
    }

    // Allocate mem on device
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));

    // Copy input matrixes to memory
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);

    addKernal<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for kernel errors
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) 
    {
        fprintf(stderr, "ERROR: Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // Kernel synchronize, checks for kernel errors
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) 
    {
        fprintf(stderr, "ERROR: Kernel synchronize failed: %d\n", cudaStatus);
        goto Error;
    }

    // Copy output matrix to memory
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);

    Error:
        hipFree(dev_c);
        hipFree(dev_a);
        hipFree(dev_b);
        
        return cudaStatus;
}

int main()
{
    const int arraySize = 5;
    const int a[arraySize] = {1, 2, 3, 4, 5};
    const int b[arraySize] = {10, 20, 30, 40, 50};
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = add(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) 
    {
        fprintf(stderr, "ERROR: Addition failed.\n");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n", c[0], c[1], c[2], c[3], c[4]);

    cudaStatus = hipDeviceReset();

    return 0;
}